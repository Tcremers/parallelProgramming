
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

int main()
{
	std::random_device rd; // obtain a random number from hardware
	std::mt19937 eng(rd()); // seed the generator
	std::uniform_int_distribution<> distr(-10, 10); // define the range

	for(int n=0; n<40; ++n)
		std::cout << distr(eng) << ' '; // generate numbers
}